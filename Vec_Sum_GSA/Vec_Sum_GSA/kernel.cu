#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <vector>
#include <chrono>
#include <algorithm>
#include <iostream>


__global__ void add(int* a, int* c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n) {
		c[index] = a[index];
	}
}
void vec_sum(int num_vec) {
	std::vector<int> vec(num_vec); // Создаем вектор размером 1 000 000
	std::fill_n(vec.begin(), vec.size(), 5);
	int sum = 0;

	auto start = std::chrono::high_resolution_clock::now();

	for (int i = 0; i < vec.size(); i++) {
		sum += vec[i];
	}

	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> diff = end - start;

	
	std::cout << "Vec: " << num_vec << std::endl << std::endl;
	
	std::cout << "CPU: " << std::endl;
	std::cout << "Sum: " << sum << std::endl;
	std::cout << "Time: " << diff.count() << " s" << std::endl << std::endl;



	int* d_a, * d_c;
	int size = vec.size() * sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, vec.data(), size, hipMemcpyHostToDevice);

	start = std::chrono::high_resolution_clock::now();

	add << <1000, 1000 >> > (d_a, d_c, vec.size());

	hipDeviceSynchronize();

	end = std::chrono::high_resolution_clock::now();
	diff = end - start;

	std::vector<int> result(vec.size());
	hipMemcpy(result.data(), d_c, size, hipMemcpyDeviceToHost);

	sum = 0;
	for (int i = 0; i < result.size(); i++) {
		sum += result[i];
	}

	std::cout << "GPU: " << std::endl;
	std::cout << "Sum: " << sum << std::endl;
	std::cout << "Time: " << diff.count() << " s" << std::endl;
	std::cout << "---------------------------------------------" << std::endl;

	hipFree(d_a);
	hipFree(d_c);

}

int main() {
	for (int i = 1000; i <= 1001000; i += 100000) {
		vec_sum(i);
	}

	return 0;
}



